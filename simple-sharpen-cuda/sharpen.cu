#include "hip/hip_runtime.h"

/**
* Sharpen
*/

#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <cstdint>
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/highgui.hpp>


using namespace cv;
using namespace std;

int video_fps;
int h_video_totalFrames;
int h_threads;
int h_width;
int h_height;

int *h_kernel;
unsigned char *h_inputVideoArray;
unsigned char *h_outputVideoArray;

VideoCapture inputVideo;
VideoWriter outputVideo;

// function aviable only on the device
__device__ void applySharpen(int rowStart,int rowEnd, unsigned char *input, unsigned char *output, int *kernel,int *w, int *h, int *totalFrames){
    int frameIndex = 0;
    for(int y=rowStart; y< rowEnd; y++){
        frameIndex = (y/((int) *h));
        int yframe = y - (frameIndex * ((int)*h));
        for(int x =0; x< *w; x++){
            int blue,green,red;
            blue=green=red=0;
            int coeficent = 0;
            for (int i = 0 ; i < 3; i++){
                for (int j = 0; j < 3; j++){
                    // para un frameIndex = a (1)   =>  row0 = h * a (1080 * 1) = 1080
                    // y = 1100   frameIndex = 1  y'= 20 = 1100 - (frameIndex * 1080)
                    int xloc = x+i;
                    int yloc = yframe+j;
                    if(0<=xloc && xloc<((int)*w)-1 && 0<=yloc && yloc<((int)*h)-1){
                        coeficent = kernel[(i*3)+j];
                        blue += input[(frameIndex* ((int)*w) *((int)*h)*3)+(xloc*((int)*h)*3)+(yloc*3)+0] *coeficent;
                        green += input[(frameIndex* ((int)*w) *((int)*h)*3)+(xloc*((int)*h)*3)+(yloc*3)+1] *coeficent;
                        red += input[(frameIndex* ((int)*w) *((int)*h)*3)+(xloc*((int)*h)*3)+(yloc*3)+2] *coeficent;
                    }
                }
            }
            output[(frameIndex*((int)*w)*((int)*h)*3)+(x*((int)*h)*3)+(yframe*3)+0] = blue;
            output[(frameIndex*((int)*w)*((int)*h)*3)+(x*((int)*h)*3)+(yframe*3)+1] = green;
            output[(frameIndex*((int)*w)*((int)*h)*3)+(x*((int)*h)*3)+(yframe*3)+2] = red;
            // printf("Sharpen applied pixel x: %d - y: %d \n", x, y);
        }
    }
}

 /**
  * CUDA Kernel Device code
  * 
  */ 
 /*****************************************************************************/
 __global__ void sharpen(unsigned char *input,unsigned char *output, int *kernel, int *totalThreads, int *width, int *height, int *totalFrames){
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    // int ini = (int)(((int) *totalFrames)/((int) *totalThreads))*(tn);
    // int fin = (int)(((int) *totalFrames)/((int) *totalThreads)) + ini;
    int ri = (int)((((int) *totalFrames) * ((int) *height))/((int) *totalThreads))*(tn);
    int rf = (int)((((int) *totalFrames) * ((int) *height))/((int) *totalThreads)) + ri;
    // printf("[%d] %d - %d \n", tn, ((int) *totalFrames), ((int) *totalThreads));
    // printf("[%d] ini: %d - fin: %d \n",tn, ri, rf);
    if(tn < (int) *totalFrames * (int) *height ){
        // printf("[%d] ini: %d - fin: %d \n",tn, ri, rf);
        applySharpen(ri, rf, input, output, kernel,width, height, totalFrames);
        // for(int i = ini; i < fin; i++){
        //     applySharpen(i, input, output, kernel,width, height);
        // }
    }
 }

 
void setVideoFrame(Mat frameInput, int frameIndex){
    for(int i=0;i<h_width;i++){
       for(int j=0;j<h_height;j++){
        h_inputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+0] = frameInput.at<Vec3b>(j,i)[0];
        h_inputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+1] = frameInput.at<Vec3b>(j,i)[1];
        h_inputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+2] = frameInput.at<Vec3b>(j,i)[2];
       }
     }
}

 void setVideoMatrix(VideoCapture inputVideo){
    Mat frameInput;
    for (int frameIndex = 0; frameIndex < h_video_totalFrames; frameIndex++){
        inputVideo.set(1,frameIndex);
        inputVideo.read(frameInput);
        if (frameInput.empty()){
            printf("Src empty");
        }
        setVideoFrame(frameInput, frameIndex);
    }
}



 /******************************************************************************
  * Host main routine
  */
 int main(int argc, char **argv)
 {   
     // define variables

     //global variabl

    Size S;

    int *d_video_totalFrames;
    int *d_threads;
    int *d_kernel;
    int *d_width;
    int *d_height;
    
    // Mat output;
    // Mat input;
    // handle errors

    hipError_t error = hipSuccess;
     //********************read parameters**********************
     if ( argc != 4 )
    {
        printf("usage: ./sharpen <Video_Path> <Video_out_Path> <ThreadsPerBlock>n");
        return -1;
    }

    String oFile = argv[2];

    //read the video and set width and height
    inputVideo = VideoCapture(argv[1]);
    if (!inputVideo.isOpened()){
        cout  << "Could not open the input video: " << argv[1] << endl;
        return -1;
    }

    //get codec type
    int ex = static_cast<int>(inputVideo.get(CAP_PROP_FOURCC));

    // get size of video
    S = Size((int) inputVideo.get(CAP_PROP_FRAME_WIDTH),
    (int) inputVideo.get(CAP_PROP_FRAME_HEIGHT));

    printf("Frame size width: %d - height: %d \n", S.width, S.height);

    video_fps = inputVideo.get(CV_CAP_PROP_FPS);
    h_video_totalFrames = 50; //inputVideo.get(7);
    h_height = S.height;
    h_width = S.width;
    
    // h_kernel = 

    // getting device information
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

   int blocks = deviceProp.multiProcessorCount;

   int h_threadsPerBlock = atoi(argv[3]);//(h_video_totalFrames*h_height)/blocks;
   h_threads = h_threadsPerBlock * blocks;
   //printf("Blocks : %d   -  threads per block %d  - TOTAL threads: %d",blocks,h_threads,blocks*h_threads);

    printf(" Processing video %s \n width: %d  - Heigh : %d \n",argv[1],h_width,h_height);

    // ************************ video pointers ***********************************
    // int *d_input;
    // int *d_output;
    unsigned char *d_inputVideoArray;
    unsigned char *d_outputVideoArray;
     // malloc and hipMalloc
     error = hipMalloc((void **)&d_height,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_height (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_kernel,sizeof(int) * 9);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_width,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_width (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_threads,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_threads (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMalloc(&d_video_totalFrames,sizeof(int));
        if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_threads (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

     error = hipMalloc(&d_inputVideoArray,h_width*h_height*sizeof(char)*3*h_video_totalFrames);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_input (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
     error = hipMalloc(&d_outputVideoArray,h_width*h_height*sizeof(char)*3*h_video_totalFrames);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_output (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    printf("hipMalloc Done");

     size_t size = h_width * h_height * 3 * sizeof(char) * h_video_totalFrames;
     h_inputVideoArray = (unsigned char *)malloc(size);
     h_outputVideoArray = (unsigned char *)malloc(size);

     // allocate kernel space

     h_kernel = (int *)malloc(sizeof(int)*9);
     h_kernel[0] = 1;
     h_kernel[1] = 0;
     h_kernel[2] = -1;
     h_kernel[3] = 0;
     h_kernel[4] = 0;
     h_kernel[5] = 0;
     h_kernel[6] = -1;
     h_kernel[7] = 0;
     h_kernel[8] = 1;

     // set initial values

     setVideoMatrix(inputVideo);

     // MemCpy: host to device

     error = hipMemcpy(d_inputVideoArray, h_inputVideoArray, size, hipMemcpyHostToDevice);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_kernel, h_kernel, sizeof(int)*9, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_threads, &h_threads, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_video_totalFrames, &h_video_totalFrames, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_width, &h_width, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_height, &h_height, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    printf("CudaMemcpy host to device done.\n");

    printf("Blocks : %d , threads per block: %d\n",blocks, h_threadsPerBlock);

    // begin clock
    clock_t begin = clock();
     // Launch kernel 
     sharpen<<<blocks,h_threadsPerBlock>>>(d_inputVideoArray,d_outputVideoArray, d_kernel, d_threads, d_width, d_height, d_video_totalFrames);

    clock_t end = clock();
    double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
    printf("Threads: %d, Time: %2.4f \n", h_threads, time_spent);

    //last error
     error = hipGetLastError();
    if (error != hipSuccess){
        fprintf(stderr, "Failed to launch sharpen (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

     // MemCpy: device to host
     error = hipMemcpy(h_outputVideoArray, d_outputVideoArray, size, hipMemcpyDeviceToHost);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy from device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


    outputVideo.open(oFile, ex, inputVideo.get(CAP_PROP_FPS), S, true);
 // fill the output video with the matrix of array frames
    Mat res;
    inputVideo.set(1,1);
    inputVideo.read(res);
    if (res.empty()){
        printf("Src empty");
    }
    for (int frameIndex = 0; frameIndex < h_video_totalFrames; frameIndex++){
        Mat it = res.clone();
        for(int i=0;i<h_width;i++){
            for(int j=0;j<h_height;j++){
                res.at<Vec3b>(j,i)[0] = h_outputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+0];
                res.at<Vec3b>(j,i)[1] = h_outputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+1];
                res.at<Vec3b>(j,i)[2] = h_outputVideoArray[(frameIndex*h_width*h_height*3)+(i*h_height*3)+(j*3)+2];
            }
        }
        outputVideo << it;
    }

     // save data


     // free memory

     hipFree(d_height);
     hipFree(d_width);
     hipFree(d_outputVideoArray);
     hipFree(d_inputVideoArray);
     hipFree(d_kernel);
     hipFree(d_threads);

     free(h_kernel);
     free(h_outputVideoArray);
     free(h_inputVideoArray);

     return 0;
 }
 
 